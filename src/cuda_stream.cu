// SPDX-License-Identifier: Apache-2.0
//
// Copyright 2025 Jasper Landa


#include <sparkplug/util/hip_stream.cuh>
#include <sparkplug/util/cuda_check.cuh>

namespace sparkplug::util {
    CudaStream::CudaStream() {
        cuda_check("failed creating CUDA Stream", hipStreamCreate, &cuda_stream_);
    }

    CudaStream::~CudaStream() {
        cuda_check("failed destroying CUDA Stream", hipStreamDestroy, cuda_stream_);
    }

    CudaStream::operator hipStream_t() const {
        return cuda_stream_;
    }

    void CudaStream::Synchronize() {
        cuda_check("CudaStream sync", hipStreamSynchronize, cuda_stream_);
    }
}