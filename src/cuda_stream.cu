#include <sparkplug/util/hip_stream.cuh>
#include <sparkplug/util/cuda_check.cuh>

namespace sparkplug::util {
    CudaStream::CudaStream() {
        cuda_check("failed creating CUDA Stream", hipStreamCreate, &cuda_stream_);
    }

    CudaStream::~CudaStream() {
        cuda_check("failed destroying CUDA Stream", hipStreamDestroy, cuda_stream_);
    }

    CudaStream::operator hipStream_t() const {
        return cuda_stream_;
    }
}