#include <sparkplug/util/device_properties.cuh>
#include <sparkplug/util/cuda_check.cuh>

namespace sparkplug::util {

    const hipDeviceProp_t& get_device_properties(){
        static const hipDeviceProp_t device_properties = [] {
            hipDeviceProp_t result{};
            cuda_check("Could not obtain Device 0 properties", hipGetDeviceProperties, &result, 0);
            return result;
        }();
        return device_properties;
    }

}