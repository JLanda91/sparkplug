// SPDX-License-Identifier: Apache-2.0
//
// Copyright 2025 Jasper Landa


#include <sstream>

#include "../include/sparkplug/util/cuda_error.cuh"

namespace sparkplug::util {
    cuda_error::cuda_error(hipError_t err, std::string_view msg)
        : std::runtime_error(ToString(err, msg))
        , code(err) {}

    std::string cuda_error::ToString(hipError_t err, std::string_view msg) {
        std::ostringstream oss{};
        oss << msg << ": " << hipGetErrorString(err) << "(Error code: " << static_cast<std::underlying_type_t<hipError_t>>(err) << ')';
        return oss.str();
    }

}
