#include <fmt/format.h>

#include "../include/sparkplug/testing/cuda_error.cuh"

namespace sparkplug::testing {
    cuda_error::cuda_error(hipError_t err, std::string_view msg)
        : std::runtime_error(fmt::format("{}: {} (Error code: {})", msg, hipGetErrorString(err), static_cast<std::underlying_type_t<hipError_t>>(err)))
        , code(err) {}
}
